#include "../include/kernel.cuh"

void run_cuBLAS(const std::vector<float>& A, const std::vector<float>& B, std::vector<float>& C, const int M, const int N, const int K, const float alpha, const float beta) {

    printf("[Kernel] Run kernal\n");

    /*** Initialize device memory ***/
    float *d_A, *d_B, *d_C;
    cudaErrChk( hipMalloc((void**)(&d_A), sizeof(float)*M*K) );
    cudaErrChk( hipMalloc((void**)(&d_B), sizeof(float)*K*N) );
    cudaErrChk( hipMalloc((void**)(&d_C), sizeof(float)*M*N) );
    cudaErrChk( hipMemcpy(d_A, A.data(), sizeof(float)*M*K, hipMemcpyHostToDevice) );
    cudaErrChk( hipMemcpy(d_B, B.data(), sizeof(float)*K*N, hipMemcpyHostToDevice) );
    cudaErrChk( hipDeviceSynchronize() )


    /*** Setup cuBLAS execution handler ***/
    hipblasHandle_t handle;
    cublasErrChk (hipblasCreate (&handle));


    /*** Run CUDA kernel ***/
    
    // Record events for performance measurement
    hipEvent_t start, stop;
    cudaErrChk( hipEventCreate(&start) );
    cudaErrChk( hipEventCreate(&stop) );
    cudaErrChk( hipEventRecord(start, NULL) );

    // Run cuBLAS kernel
    cublasErrChk( hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N) );

    // End of events
    cudaErrChk(hipEventRecord(stop, NULL));
    cudaErrChk(hipEventSynchronize(stop));
    float msec_total = 0.0f;
    float gflo = 2.0f*M*N*K*1e-9; // multiply and add
    cudaErrChk(hipEventElapsedTime(&msec_total, start, stop));
    printf(" -- elaped time: %.4f sec\n", msec_total*1e-3);
    printf(" -- gFlops : %.4f gflops\n", gflo/(msec_total*1e-3));

    cudaErrChk( hipMemcpy(C.data(), d_C, M*N*sizeof(float), hipMemcpyDeviceToHost) );
    cudaErrChk( hipDeviceSynchronize() )


    /*** Finalize ***/
    cudaErrChk( hipFree(d_A) );
    cudaErrChk( hipFree(d_B) );
    cudaErrChk( hipFree(d_C) );
    cublasErrChk( hipblasDestroy(handle) );
    
}
